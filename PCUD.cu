#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <float.h>

#define SWARM_SIZE 30
#define MAX_ITER 100
#define DIMENSIONS 2
#define X_MAX 100
#define X_MIN -100
#define V_MAX 20

struct Particle {
    double position[DIMENSIONS];
    double velocity[DIMENSIONS];
    double personal_best_position[DIMENSIONS];
    double personal_best_value;
};

__device__ double fitness_function(double *position) {
    double sum = 0;
    for (int i = 0; i < DIMENSIONS; i++) {
        sum += position[i] * position[i];
    }
    return sum;
}

__global__ void init_particles(Particle *swarm, hiprandState *states, int seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= SWARM_SIZE) return;

    hiprand_init(seed, idx, 0, &states[idx]);
    for (int d = 0; d < DIMENSIONS; d++) {
        swarm[idx].position[d] = X_MIN + hiprand_uniform(&states[idx]) * (X_MAX - X_MIN);
        swarm[idx].velocity[d] = 0.5 * V_MAX * (hiprand_uniform(&states[idx]) * 2 - 1);
        swarm[idx].personal_best_position[d] = swarm[idx].position[d];
    }
    swarm[idx].personal_best_value = fitness_function(swarm[idx].personal_best_position);
}

__global__ void update_particles(Particle *swarm, double *global_best_position, hiprandState *states) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= SWARM_SIZE) return;

    double r1 = hiprand_uniform(&states[idx]);
    double r2 = hiprand_uniform(&states[idx]);

    for (int d = 0; d < DIMENSIONS; d++) {
        swarm[idx].velocity[d] = 0.5 * swarm[idx].velocity[d]
                                 + 2.0 * r1 * (swarm[idx].personal_best_position[d] - swarm[idx].position[d])
                                 + 2.0 * r2 * (global_best_position[d] - swarm[idx].position[d]);

        if (swarm[idx].velocity[d] > V_MAX) swarm[idx].velocity[d] = V_MAX;
        if (swarm[idx].velocity[d] < -V_MAX) swarm[idx].velocity[d] = -V_MAX;

        swarm[idx].position[d] += swarm[idx].velocity[d];
        if (swarm[idx].position[d] > X_MAX) swarm[idx].position[d] = X_MAX;
        if (swarm[idx].position[d] < X_MIN) swarm[idx].position[d] = X_MIN;
    }

    double current_value = fitness_function(swarm[idx].position);
    if (current_value < swarm[idx].personal_best_value) {
        swarm[idx].personal_best_value = current_value;
        for (int d = 0; d < DIMENSIONS; d++) {
            swarm[idx].personal_best_position[d] = swarm[idx].position[d];
        }
    }
}

int main() {
    Particle *d_swarm;
    double *d_global_best_position, global_best_position[DIMENSIONS];
    double *d_global_best_value, global_best_value = DBL_MAX;
    hiprandState *d_states;

    hipMalloc(&d_swarm, sizeof(Particle) * SWARM_SIZE);
    hipMalloc(&d_global_best_position, sizeof(double) * DIMENSIONS);
    hipMalloc(&d_global_best_value, sizeof(double));
    hipMalloc(&d_states, sizeof(hiprandState) * SWARM_SIZE);

    // Prepare CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;

    // Start timing after initialization
    hipEventRecord(start);

    dim3 blocks((SWARM_SIZE + 15) / 16);
    dim3 threads(16);

    init_particles<<<blocks, threads>>>(d_swarm, d_states, time(NULL));
    hipDeviceSynchronize();

    for (int iter = 0; iter < MAX_ITER; iter++) {
        update_particles<<<blocks, threads>>>(d_swarm, d_global_best_position, d_states);
        hipDeviceSynchronize(); // Ensure all updates are completed
    }

    // Stop timing after computation is done
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    // Copy the global best position and value back to host
    hipMemcpy(global_best_position, d_global_best_position, sizeof(double) * DIMENSIONS, hipMemcpyDeviceToHost);
    hipMemcpy(&global_best_value, d_global_best_value, sizeof(double), hipMemcpyDeviceToHost);

    printf("Best Value = %f at position (%f, %f)\n", global_best_value, global_best_position[0], global_best_position[1]);


    // Cleanup
    hipFree(d_swarm);
    hipFree(d_global_best_position);
    hipFree(d_global_best_value);
    hipFree(d_states);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
